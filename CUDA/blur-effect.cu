#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <pthread.h>
#include <string.h>
#include <stdio.h>
#include <opencv/cvaux.h>
#include <opencv2/opencv.hpp>
#include <opencv/highgui.h>
#include <opencv/cxcore.h>
#include <opencv/cv.h>
#include <opencv2/imgproc/imgproc.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <string>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
struct pixel{
	int R,G,B;
};
__global__ void blur(const pixel *src_img, pixel *dst_img,int Ksize, int h, int w, int numElements, int threads){


	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
        if (index<w-Ksize)
    {
	int n;
	int l;
	int i;
	int j;
    int sumR=0;
    int sumG=0;
    int sumB=0;
    int klimitr;	//limites de convolucion
	int klimitc;	//limites de convolucion  
    int aux=((h-(2*(Ksize/2)))/threads);
    int start= index*aux;
    int end=start+aux;
    int Km=Ksize/2;

if (threads>(h-(2*Km)))
        {
             aux=1;
        }

		//i=index+(Ksize/2);
		for (i = start+Ksize/2; i < end+Ksize/2; ++i)
        {
            /* code */
        
	  	for (j = Ksize/2; j < w-Ksize/2; ++j)
	  	{
	  		sumB=0;
	  		sumG=0;
	  		sumR=0;
	  		
	  		if (Ksize%2==0)
	  		{
	  			klimitr=i+(Ksize/2)-1;
	  			klimitc=j+(Ksize/2)-1;
	  		}else{
                klimitr=i+(Ksize/2);    //limites de convolucion
                klimitc=j+(Ksize/2);    //limites de convolucion
            }
	  		for (l = i-(Ksize/2); l <= klimitr; ++l)		//convocucion
	  		{
	  			for (n = j-(Ksize/2); n <= klimitc; ++n)	//convolucion
	  			{
	  				sumR+=src_img[l*w+n].R;
	                sumG+=src_img[l*w+n].G;
	                sumB+=src_img[l*w+n].B;		//canal rojo suma de vecinos
	  			}
	  		}
	  		sumR-=src_img[i*w+j].R;
	        sumG-=src_img[i*w+j].G;
	        sumB-=src_img[i*w+j].B;			//canal rojo resta de px a tratar
	  		
	  		dst_img[i*w+j].R=sumR/((Ksize*Ksize)-1);
	        dst_img[i*w+j].G=sumG/((Ksize*Ksize)-1);
	        dst_img[i*w+j].B=sumB/((Ksize*Ksize)-1);
	  		}
	  	}
  	}
  }








using namespace cv;

int main(int argc, char const *argv[])
{
	Mat mat = imread(argv[1], CV_LOAD_IMAGE_COLOR);
	int Ksize=atoi(argv[2]);
	int h=mat.rows;
	int w=mat.cols;
	
/*  if (!img)
  {
    printf("Image: can NOT Load!!!\n");
    return 1;
  }*/
  copyMakeBorder( mat, mat, Ksize/2, Ksize/2, Ksize/2, Ksize/2, BORDER_REPLICATE);
  	int tmpH=mat.rows;
	int tmpW=mat.cols;
  	hipError_t err = hipSuccess;
	int numElements = tmpH*tmpW;
	int size = numElements*sizeof(pixel);
	
	pixel * h_img = (pixel*)malloc(size);



	if (h_img == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }


  for (int i = 0; i < tmpH; ++i)
  {
	for (int j = 0; j < tmpW; ++j)
	{
	  h_img[(tmpW*i)+j].B=(mat.at<cv::Vec3b>(i,j)[0]);
	  h_img[(tmpW*i)+j].G=(mat.at<cv::Vec3b>(i,j)[1]);
	  h_img[(tmpW*i)+j].R=(mat.at<cv::Vec3b>(i,j)[2]);
    }
}



	pixel * d_img;
	err = hipMalloc((void**)&d_img, size);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_img (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    pixel * d_imgDst;

	err = hipMalloc((void**)&d_imgDst, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector D_imgDst (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    err = hipMemcpy(d_img, h_img, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_R from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

 

    int threadsPerBlock = atoi(argv[3]);
    int blocks=atoi(argv[4]);
    if (blocks==0){
    	blocks= (h/threadsPerBlock)+1;
    }
    int threads=blocks*threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threadsPerBlock);
    blur<<<blocks,threadsPerBlock>>>(d_img, d_imgDst, Ksize, tmpH, tmpW, numElements, threads);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch blur kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_img, d_imgDst, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_dstR from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_img);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_img (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_imgDst);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_imgDst (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    /*
	implementacion de vectores resultantes de cuda blur kernel a Mat m
    */
    for (int i = 0; i < tmpH; ++i)
        {
        for (int j = 0; j < tmpW; ++j)
            {
            (mat.at<cv::Vec3b>(i,j)[0])=h_img[(tmpW*i)+j].B;
            (mat.at<cv::Vec3b>(i,j)[1])=h_img[(tmpW*i)+j].G;
            (mat.at<cv::Vec3b>(i,j)[2])=h_img[(tmpW*i)+j].R;

        }
    }

    Mat cropedImage = mat(Rect(Ksize/2,Ksize/2,w,h));
    std::string filename=argv[1];
    filename=filename.substr(0,sizeof(argv[1]));
    cv::imwrite( filename+" --NOCROPPED-- .jpg", mat);
    cv::imwrite( filename+" --blurred-- .jpg", cropedImage);

    free(h_img);


    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;

}